#include "hip/hip_runtime.h"
#include <stdio.h>

#include "../common/book.h"

#define N 10000

__global__ void add(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate memory on GPU
    HandleError(hipMalloc(&dev_a, N * sizeof(int)), "hipMalloc dev_a", 20);
    HandleError(hipMalloc(&dev_b, N * sizeof(int)), "hipMalloc dev_b", 21);
    HandleError(hipMalloc(&dev_c, N * sizeof(int)), "hipMalloc dev_c", 22);

    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i * i * i;
    }

    // copy data
    HandleError(hipMemcpy(dev_a,
                           a,
                           N * sizeof(int),
                           hipMemcpyHostToDevice),
                "hipMemcpy dev_a", 29);
    HandleError(hipMemcpy(dev_b,
                           b,
                           N * sizeof(int),
                           hipMemcpyHostToDevice),
                "hipMemcpy dev_b", 34);

    add<<<(N + 127)/128,128>>>(dev_a, dev_b, dev_c);

    // copy result back to host
    HandleError(hipMemcpy(c,
                           dev_c,
                           N * sizeof(int),
                           hipMemcpyDeviceToHost),
                "hipMemcpy to host dev_c", 43);

    // display results
    for (int i=0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    HandleError(hipFree(dev_a), "hipFree dev_a", 54);
    HandleError(hipFree(dev_b), "hipFree dev_b", 55);
    HandleError(hipFree(dev_c), "hipFree dev_c", 56);
    //
    return 0;
}
